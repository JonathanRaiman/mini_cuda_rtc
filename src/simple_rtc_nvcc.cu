#include "hip/hip_runtime.h"
#include "config.h"
#include "array.h"

#include <cstdlib>      // EXIT_FAILURE, etc
#include <string>
#include <iostream>
#include <fstream>
#include <dlfcn.h>      // dynamic library loading, dlopen() etc
#include <cxxabi.h>
#include <sys/stat.h>
#include <unordered_map>
#include <tuple>

#define STR(x) __THIS_IS_VERY_ABNOXIOUS(x)
#define __THIS_IS_VERY_ABNOXIOUS(tok) #tok

template<typename Cls>
std::string get_class_name() {
    int status;
    char * demangled = abi::__cxa_demangle(
        typeid(Cls).name(),
        0,
        0,
        &status
    );
    return std::string(demangled);
}

template<typename... Args, typename std::enable_if<sizeof... (Args) == 0, int>::type = 0>
void get_function_arguments(int i, std::string* call_ptr) {}

template<typename Arg, typename... Args>
void get_function_arguments(int i, std::string* call_ptr) {
    std::string& call = *call_ptr;
    if (i > 0) {
        call = call + ", ";
    }
    call = call + get_class_name<Arg>() + " " + (char)(((int)'a') + i);
    get_function_arguments<Args...>(i+1, call_ptr);
}

template<typename... Args>
std::string get_function_arguments() {
    std::string s;
    get_function_arguments<Args...>(0, &s);
    return s;
}

std::string get_call_args(std::size_t num_args) {
    std::string call_args;
    for (int i = 0; i < num_args; i++) {
        if (i > 0) {
            call_args = call_args + ", ";
        }
        call_args = call_args + (char)(((int)'a') + i);
    }
    return call_args;
}

struct ModulePointer {
    void* module_;
    std::string libname_;

    ModulePointer(const std::string& libname) : module_(NULL), libname_(libname) {
        module_ = dlopen(libname_.c_str(), RTLD_LAZY);
        if(!module_) {
            std::cerr << "error loading library:\n" << dlerror() << std::endl;
            exit(EXIT_FAILURE);
        }
    }

    ~ModulePointer() {
        if (module_) {
            dlclose(module_);
        }
    }
};

struct Module {
    std::shared_ptr<ModulePointer> module_ptr_;

    Module() : module_ptr_(NULL) {}
    Module(const std::string& libname) :
            module_ptr_(std::make_shared<ModulePointer>(libname)) {
    }

    void* module() {
        return module_ptr_->module_;
    }

    template<typename T>
    T get_symbol(const std::string& name) {
        void* symbol = dlsym(module(), name.c_str());
        const char* dlsym_error = dlerror();
        if (dlsym_error != NULL) {
            std::cerr << "error loading symbol:\n" << dlsym_error << std::endl;
            exit(EXIT_FAILURE);
        }
        return reinterpret_cast<T>(symbol);
    }

};

namespace std {
    template<typename... TTypes>
    class hash<std::tuple<TTypes...>> {
        private:
            typedef std::tuple<TTypes...> Tuple;

            template<int N>
            size_t operator()(Tuple value) const {
                return 0;
            }

            template<int N, typename THead, typename... TTail>
            size_t operator()(Tuple value) const {
                constexpr int Index = N - sizeof...(TTail) - 1;
                return hash<THead>()(std::get<Index>(value)) ^ operator()<N, TTail...>(value);
            }

        public:
            size_t operator()(Tuple value) const {
                return operator()<sizeof...(TTypes), TTypes...>(value);
            }
    };
}

bool file_exists (const std::string& fname) {
    struct stat buffer;
    return (stat (fname.c_str(), &buffer) == 0);
}

struct Headerfile {
    std::string path_;
    std::string name_;
    Headerfile(const std::string& path, const std::string& name) :
        path_(path), name_(name) {}
};

struct Compiler {
    std::vector<Headerfile> headerfiles_;
    std::string outpath_;
    std::unordered_map<std::tuple<std::size_t, std::size_t>, Module> modules;

    Compiler(const std::vector<Headerfile>& headerfiles, const std::string& outpath)
            : headerfiles_(headerfiles), outpath_(outpath) {
        copy_headers();
    }

    void copy_headers() const {
        for (auto& header : headerfiles_) {
            system(
                make_message("cp ", header.path_, " ", outpath_).c_str()
            );
        }
    }

    std::string header_file_includes() const {
        std::stringstream ss;
        for (auto& header : headerfiles_) {
            ss << "#include \"", header.name_, "\"\n";
        }
        return ss.str();
    }

    template<typename... Args>
    void write_code(const std::string& fname,
                    const std::string& code,
                    const std::string& funcname) {
        std::ofstream out(fname.c_str(), std::ofstream::out);
        if (out.bad()) {
            std::cout << "cannot open " << fname << std::endl;
            exit(EXIT_FAILURE);
        }
        // add header to code (and extern c to avoid name mangling)
        std::string newcode = make_message(
            header_file_includes(),
            code, "\n", "extern \"C\" void maker (",
            get_function_arguments<Args...>(),
            "){\n", funcname, '(', get_call_args(sizeof...(Args)), ");}"
        );
        out << newcode;
        out.flush();
        out.close();
    }

    bool compile_code(const std::string& source,
                      const std::string& dest,
                      const std::string& logfile) {
        std::string cmd = "nvcc -std=c++11 " + source + " -o " + dest
                          + " -O2 -shared &> " + logfile;
        int ret = system(cmd.c_str());
        return WEXITSTATUS(ret) == EXIT_SUCCESS;
    }

    template<typename... Args>
    void create_module(const std::string& save_name,
                       const std::string& code,
                       const std::string& funcname,
                       bool force_recompilation,
                       const std::tuple<std::size_t, std::size_t>& module_key) {
        std::string libfile = save_name + ".so";
        bool module_never_compiled = !file_exists(libfile);
        if (force_recompilation || module_never_compiled) {
            std::cout << "Compiling..." << std::endl;
            std::string cppfile = save_name + ".cu";
            std::string logfile = save_name + ".log";

            write_code<Args...>(cppfile, code, funcname);
            bool success = compile_code(
                cppfile,
                libfile,
                logfile
            );

            if (!success) {
                std::cout << "Compilation failed, see " << logfile << std::endl;
                exit(EXIT_FAILURE);
            }
        } else {
            std::cout << "Module previously compiled: reusing." << std::endl;
        }
        Module module(libfile);
        modules[module_key] = module;
    }

    // compile code, instantiate class and return pointer to base class
    // https://www.linuxjournal.com/article/3687
    // http://www.tldp.org/HOWTO/C++-dlopen/thesolution.html
    // https://stackoverflow.com/questions/11016078/
    // https://stackoverflow.com/questions/10564670/
    template<typename... Args>
    std::function<void(Args...)> compile(
            const std::string& code,
            std::string funcname,
            bool force_recompilation) {

        std::size_t code_hash = std::hash<std::string>()(code);
        std::string func_args = get_function_arguments<Args...>();
        std::size_t arg_hash  = std::hash<std::string>()(func_args);
        std::tuple<std::size_t, std::size_t> module_key(code_hash, arg_hash);

        bool module_never_loaded = modules.find(module_key) == modules.end();

        if (force_recompilation || module_never_loaded) {
            create_module<Args...>(
                make_message(outpath_, "/", code_hash, arg_hash),
                code, funcname,
                force_recompilation,
                module_key
            );
        } else {
            std::cout << "Module previously loaded: reusing." << std::endl;
        }
        std::function<void(Args...)> method = modules[module_key].get_symbol<void(*)(Args...)>("maker");
        return method;
    }
};

std::function<void(ArrayGather<float>, Array<float>)> get_func_with_operator(
        Compiler& compiler,
        std::string operator_name) {
    std::string code = (
        "struct CustomSaver {\n"
        "    template<typename T>\n"
        "    static void XINLINE save(T& left, const T& right) {\n"
        "        left " + operator_name + "= right;\n"
        "    }\n"
        "};\n"
        "\n"
        "template<typename SrcT>\n"
        "void rtc_func(ArrayGather<SrcT> source,\n"
        "              const Array<SrcT>& updates,\n"
        "              hipStream_t stream = NULL) {\n"
        "    scatter_saver<CustomSaver>(\n"
        "        source.array_, source.indices_, updates, stream\n"
        "    );\n"
        "};"
    );
    return compiler.compile<ArrayGather<float>, Array<float>>(code, "rtc_func", false);
}

int main(int argc, char** argv) {
    int dim = 5;
    int cols = 3;

    Array<float> source({dim, cols}, DEVICE_T_GPU);
    source = 0;

    Array<float> updates({dim, cols}, DEVICE_T_GPU);
    updates = 2;

    Array<int> indices({dim}, DEVICE_T_CPU);
    int i = 0;
    for (auto index : {0, 0, 2, 1, 2}) {
        indices[i++] = index;
    }
    indices.print();
    auto indices_gpu = indices.to_device(DEVICE_T_GPU);
    source.print();
    // increment repeatedly at this location:
    source[indices_gpu] += updates;
    source.print();
    // decrement repeatedly at this location:
    source[indices_gpu] -= updates;
    source.print();
    // not well defined in many to one setup
    source[indices_gpu] = updates;
    source.print();

    // RTC
    std::string operator_name = "+";
    // choose symbol at runtime
    if (argc > 1) {
        operator_name = argv[1];
    }

    Compiler compiler(
        {Headerfile(STR(PROJECT_DIR) "/src/array.h", "array.h")},
        "/tmp"
    );

    // run functor defined by user at runtime:
    auto func = get_func_with_operator(compiler, operator_name);
    func(source[indices_gpu], updates);
    source.print();

    // decrement repeatedly at this location:
    auto func2 = get_func_with_operator(compiler, "-");
    func2(source[indices_gpu], updates);
    source.print();

    // run functor defined by user at again:
    func(source[indices_gpu], updates);
    source.print();

    return EXIT_SUCCESS;
}
