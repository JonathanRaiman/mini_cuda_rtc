#include "device.h"
#include <stdexcept>
#include <iostream>
#include "make_message.h"

void Device::set_gpu_device(int device) {
    if (gpu_device != device) {
        int count;
        hipGetDeviceCount(&count);
        if (count == 0) {
            throw std::runtime_error(
                "no gpu devices found"
            );
        } else {
            std::cout << count << " gpu device" << (count == 1 ? " " : "s ") << "found" << std::endl;
        }

        auto status = hipSetDevice(device);
        if (status != hipSuccess) {
            throw std::runtime_error(
                make_message(
                    "could not set the gpu device to ",
                    device, ", reason = ", hipGetErrorString(status)
                )
            );
        }
        gpu_device = device;
    }
}

int Device::gpu_device = -1;
